#include "hip/hip_runtime.h"
#include <DO/Sara/Core/DebugUtilities.hpp>

#include <DO/Shakti/MultiArray/Cuda/Array.hpp>
#include <DO/Shakti/Utilities/Timer.hpp>

#include "ImageProcessing.hpp"

#include "image_processing.hpp"


using namespace std;


namespace DO { namespace Shakti {

  void apply_row_based_convolution(float *out, const float *in, const float *kernel,
                                   int kernel_size, const int *sizes)
  {
    const dim3 block_size{ 16, 16 };
    const dim3 grid_size{
      (sizes[0] + block_size.x - 1) / block_size.x,
                      (sizes[1] + block_size.y - 1) / block_size.y
    };

    tic();
    Cuda::Array<float> in_array{ in, { sizes[0], sizes[1] } };
    MultiArray<float, 2> out_array{ { sizes[0], sizes[1] } };
    toc("Host to device transfer");

    tic();
    CHECK_CUDA_RUNTIME_ERROR(hipBindTextureToArray(in_texture, in_array));
    hipMemcpyToSymbol(HIP_SYMBOL(convolution_kernel), kernel, sizeof(float) * kernel_size);
    hipMemcpyToSymbol(HIP_SYMBOL(convolution_kernel_size), &kernel_size, sizeof(int));
    apply_row_based_convolution<<<grid_size, block_size>>>(out_array.data());
    CHECK_CUDA_RUNTIME_ERROR(hipUnbindTexture(in_texture));
    toc("Row based convolution");

    tic();
    out_array.copy_to_host(out);
    toc("Device to host transfer");
  }

  void apply_column_based_convolution(float *out, const float *in, const float *kernel,
                                      int kernel_size, const int *sizes)
  {
    const dim3 block_size{ 16, 16 };
    const dim3 grid_size{
      (sizes[0] + block_size.x - 1) / block_size.x,
      (sizes[1] + block_size.y - 1) / block_size.y
    };

    tic();
    Cuda::Array<float> in_array{ in, { sizes[0], sizes[1] } };
    MultiArray<float, 2> out_array{ { sizes[0], sizes[1] } };
    toc("Host to device transfer");

    tic();
    CHECK_CUDA_RUNTIME_ERROR(hipBindTextureToArray(in_texture, in_array));
    CHECK_CUDA_RUNTIME_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(convolution_kernel), kernel, sizeof(float) * kernel_size));
    CHECK_CUDA_RUNTIME_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(convolution_kernel_size), &kernel_size, sizeof(int)));
    apply_column_based_convolution<<<grid_size, block_size>>>(out_array.data());
    CHECK_CUDA_RUNTIME_ERROR(hipUnbindTexture(in_texture));
    toc("Column based convolution");

    tic();
    out_array.copy_to_host(out);
    toc("Device to host transfer");
  }

  void apply_x_derivative(float *out, const float *in, const int *sizes)
  {
    constexpr float kernel[] = { -1.f, 0.f, 1.f };
    constexpr int kernel_size{ 3 };
    apply_column_based_convolution(out, in, kernel, kernel_size, sizes);
  }

  void apply_y_derivative(float *out, const float *in, const int *sizes)
  {
    constexpr float kernel[] = { -1.f, 0.f, 1.f };
    constexpr int kernel_size{ 3 };
    apply_row_based_convolution(out, in, kernel, kernel_size, sizes);
  }

} /* namespace Shakti */
} /* namespace DO */


namespace DO { namespace Shakti {

  void GaussianFilter::set_sigma(float sigma)
  {
    auto kernel_size = static_cast<int>(2.f * _truncation_factor * sigma + 1.f);
    kernel_size = std::max(3, kernel_size);
    if (kernel_size % 2 == 0)
      ++kernel_size;

    auto sum = float{ 0.f };
    _kernel.resize(kernel_size);
    for (auto i = int{ 0 }; i < kernel_size; ++i)
    {
      auto x = i - kernel_size/2;
      _kernel[i] = exp(-x*x / (2.f*sigma*sigma));
      sum += _kernel[i];
    }
    for (auto i = int{ 0 }; i < kernel_size; ++i)
      _kernel[i] /= sum;

    CHECK_CUDA_RUNTIME_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(
      convolution_kernel), _kernel.data(), sizeof(float) * _kernel.size()));
    CHECK_CUDA_RUNTIME_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(
      convolution_kernel_size), &kernel_size, sizeof(int)));
  }

  void GaussianFilter::operator()(float *out, const float *in, const int *sizes) const
  {
    const dim3 block_size{ 16, 16 };
    const dim3 grid_size{
      (sizes[0] + block_size.x - 1) / block_size.x,
      (sizes[1] + block_size.y - 1) / block_size.y
    };

    tic();
    Cuda::Array<float> in_array{ in, { sizes[0], sizes[1] } };
    MultiArray<float, 2> out_array{ { sizes[0], sizes[1] } };
    toc("Host to device transfer");

    tic();
    {
      CHECK_CUDA_RUNTIME_ERROR(hipBindTextureToArray(in_texture, in_array));
      apply_column_based_convolution<<<grid_size, block_size>>>(out_array.data());
      in_array.copy_from(out_array.data(), out_array.sizes(), hipMemcpyDeviceToDevice);
      apply_row_based_convolution<<<grid_size, block_size>>>(out_array.data());

      CHECK_CUDA_RUNTIME_ERROR(hipUnbindTexture(in_texture));
    }
    toc("Gaussian filter");

    tic();
    out_array.copy_to_host(out);
    toc("Device to host transfer");
  }

} /* namespace Shakti */
} /* namespace DO */
